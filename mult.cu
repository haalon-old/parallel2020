#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "problem.h"
#include "block.hpp"

#define REDUCTION_BLOCK_SIZE 1024
#define THREADSINBLOCK 512
#define THREAD_AXIS 8

__constant__ int N_D;

__constant__ double C_X_D;
__constant__ double C_Y_D;
__constant__ double C_Z_D;

__constant__ int sx_d, ex_d, nx_d;
__constant__ int sy_d, ey_d, ny_d;
__constant__ int sz_d, ez_d, nz_d;

__constant__ double H_X_D, H_Y_D, H_Z_D, TAU_D;
__constant__ double L_X_D, L_Y_D, L_Z_D;

__constant__ double * prev_d;
__constant__ double * curr_d;
__constant__ double * next_d;

__constant__ double * edges_d[6];
__constant__ double * new_edges_d[6];

size_t edge_sizes[6];

__device__ double u_analytical_d(double l_x, double l_y, double l_z, double x, double y, double z, double t)
{
    double a_t = PI * sqrt(1/(l_x*l_x) + 4/(l_y*l_y) + 9/(l_z*l_z));
    return sin(PI*x/l_x) * sin(2*PI*y/l_y) * sin(3*PI*z/l_z) * cos(a_t*t);
}

#define SAFE_CALL( CallInstruction ) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
        printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA API function, aborting..."; \
    } \
}


#define SAFE_KERNEL_CALL( KernelCallInstruction ){ \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel execution, aborting..."; \
    } \
}

extern void initDevice(Block * b){
    // set extern consts
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(N_D), &N, sizeof(int)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(C_X_D), &C_X, sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(C_Y_D), &C_Y, sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(C_Z_D), &C_Z, sizeof(double)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(H_X_D), &H_X, sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(H_Y_D), &H_Y, sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(H_Z_D), &H_Z, sizeof(double)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(L_X_D), &L_X, sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(L_Y_D), &L_Y, sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(L_Z_D), &L_Z, sizeof(double)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(TAU_D), &TAU, sizeof(double)));

    // set block parameters
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sx_d), &b->sx, sizeof(int)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ex_d), &b->ex, sizeof(int)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nx_d), &b->nx, sizeof(int)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sy_d), &b->sy, sizeof(int)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ey_d), &b->ey, sizeof(int)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ny_d), &b->ny, sizeof(int)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sz_d), &b->sz, sizeof(int)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ez_d), &b->ez, sizeof(int)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nz_d), &b->nz, sizeof(int)));


    // set arrays
    int size = b->nx * b->ny * b->nz;
    int sizeX = b->ny * b->nz, sizeY = b->nx * b->nz, sizeZ = b->nx * b->ny;
    edge_sizes[0] = sizeX; edge_sizes[1] = sizeY; edge_sizes[2] = sizeZ;
    edge_sizes[3] = sizeZ; edge_sizes[4] = sizeY; edge_sizes[5] = sizeX;
    double * temp_host;

    // 1) allocate memory, temp_host will have pointer to dev memory
    // (not needed anymore!) 2) copy host data to device data 
    // 3) copy the pointer itself to the device
    SAFE_CALL(hipMalloc((void**)&temp_host, size * sizeof(double)));       
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(prev_d), &temp_host, sizeof(double *)));

    SAFE_CALL(hipMalloc((void**)&temp_host, size * sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(curr_d), &temp_host, sizeof(double *)));

    SAFE_CALL(hipMalloc((void**)&temp_host, size * sizeof(double)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(next_d), &temp_host, sizeof(double *)));

    double * host_edges_temp[6];

    for (int i = 0; i < 6; ++i) 
        SAFE_CALL(hipMalloc((void**)&host_edges_temp[i], edge_sizes[i] * sizeof(double)));
    
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges_d), &host_edges_temp, 6 * sizeof(double *)));

    for (int i = 0; i < 6; ++i) 
        SAFE_CALL(hipMalloc((void**)&host_edges_temp[i], edge_sizes[i] * sizeof(double)));
    
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(new_edges_d), &host_edges_temp, 6 * sizeof(double *)));
}

void swap() {
    double * next, curr, prev;
    SAFE_CALL(hipMemcpyFromSymbol(&next, HIP_SYMBOL(next_d), sizeof(double *)));
    SAFE_CALL(hipMemcpyFromSymbol(&curr, HIP_SYMBOL(curr_d), sizeof(double *)));
    SAFE_CALL(hipMemcpyFromSymbol(&prev, HIP_SYMBOL(prev_d), sizeof(double *)));

    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(curr_d), &next, sizeof(double *)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(prev_d), &curr, sizeof(double *)));
    SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(next_d), &prev, sizeof(double *)));
}

__device__ char onConstEdge_D(int i, int j, int k) {
    if(!PERIOD_X && (i==0 || i==N_D))
        return 1;

    if(!PERIOD_Y && (j==0 || j==N_D))
        return 1;

    if(!PERIOD_Z && (k==0 || k==N_D))
        return 1;

    return 0;
}

__device__ double& get(double * layer,  int i, int j, int k)
{
    if(i==sx_d-1)
        return edges_d[0][nz_d*(j - sy_d) + (k - sz_d)];

    if(i==ex_d+1)
        return edges_d[5][nz_d*(j - sy_d) + (k - sz_d)];

    if(j==sy_d-1)
        return edges_d[1][nz_d*(i - sx_d) + (k - sz_d)];

    if(j==ey_d+1)
        return edges_d[4][nz_d*(i - sx_d) + (k - sz_d)];

    if(k==sz_d-1)
        return edges_d[2][ny_d*(i - sx_d) + (j - sy_d)];

    if(k==ez_d+1)
        return edges_d[3][ny_d*(i - sx_d) + (j - sy_d)];


    return layer[nz_d*ny_d*(i - sx_d) + nz_d*(j - sy_d) + (k - sz_d)];
}

__device__ void setNewEdges(double val, int i, int j, int k) {
    if(i==sx_d)
        new_edges_d[0][nz_d*(j - sy_d) + (k - sz_d)] = val;

    if(i==ex_d)
        new_edges_d[5][nz_d*(j - sy_d) + (k - sz_d)] = val;

    if(j==sy_d)
        new_edges_d[1][nz_d*(i - sx_d) + (k - sz_d)] = val;

    if(j==ey_d)
        new_edges_d[4][nz_d*(i - sx_d) + (k - sz_d)] = val;

    if(k==sz_d)
        new_edges_d[2][ny_d*(i - sx_d) + (j - sy_d)] = val;

    if(k==ez_d)
        new_edges_d[3][ny_d*(i - sx_d) + (j - sy_d)] = val;
}

__device__ double delta(double * curr, int i, int j, int k) {
    double d_x, d_y, d_z;

    d_x = (get(curr, i+1, j, k) - get(curr, i, j, k)) * C_X_D + (get(curr, i-1, j, k) - get(curr, i, j, k)) * C_X_D;
    d_y = (get(curr, i, j-1, k) - get(curr, i, j, k)) * C_Y_D + (get(curr, i, j+1, k) - get(curr, i, j, k)) * C_Y_D;
    d_z = (get(curr, i, j, k-1) - get(curr, i, j, k)) * C_Z_D + (get(curr, i, j, k+1) - get(curr, i, j, k)) * C_Z_D;

    return d_x*C_X_D + d_y*C_Y_D + d_z*C_Z_D;
}



__global__ void __calc_n__(int num) {
    int gridOffsetX = blockDim.x * blockIdx.x;
    int gridOffsetY = blockDim.y * blockIdx.y;
    int gridOffsetZ = blockDim.z * blockIdx.z;

    // shift by s(.)_d, so i j k are indexing from the start of the block
    int i = threadIdx.x + gridOffsetX + sx_d;
    int j = threadIdx.y + gridOffsetY + sy_d;
    int k = threadIdx.z + gridOffsetZ + sz_d;

    // it shouldnt really happen, unless we use weird grid sizes
    if(i > ex_d || j > ey_d || k > ez_d)
        return;

    double val;

    switch(num) {
        case  0: val = u_analytical_d(L_X_D, L_Y_D, L_Z_D, H_X_D*i, H_Y_D*j, H_Z_D*k, 0); break;
        case  1: val = onConstEdge_D(i,j,k) ? 0 : get(curr_d, i, j, k) + delta(curr_d, i,j,k)/2.0; break;
        default: val = onConstEdge_D(i,j,k) ? 0 : 2*get(curr_d, i, j, k) + delta(curr_d, i, j, k) - get(prev_d, i, j, k); break;
    }

    get(next_d, i,j,k) = val;
    setNewEdges(val, i,j,k);

}

extern void launch_calc(Block * b, int num)
{
    double * temp_edges[6];

    // don't needed on the zeroth step
    if(num) {
        // same place as in block
        swap();

        // copy edges we recieved to the gpu
        SAFE_CALL(hipMemcpyFromSymbol(&temp_edges, HIP_SYMBOL(edges_d), 6 * sizeof(double *)));
        for (int i = 0; i < 6; ++i) {
            SAFE_CALL(hipMemcpy((void*)temp_edges[i], (void*)b->edges[i], edge_sizes[i] * sizeof(double), hipMemcpyHostToDevice));
        }
    }

    dim3 blockDim = dim3(THREAD_AXIS,THREAD_AXIS,THREAD_AXIS);
    int gridDimX = (b->nx - 1)/THREAD_AXIS + 1;
    int gridDimY = (b->ny - 1)/THREAD_AXIS + 1;
    int gridDimZ = (b->nz - 1)/THREAD_AXIS + 1;
    dim3 gridDim = dim3(gridDimX, gridDimY, gridDimZ);

    // get pointer to next array on device, and put it into temp
    SAFE_CALL(hipMemcpyFromSymbol(&temp_edges, HIP_SYMBOL(new_edges_d), 6 * sizeof(double *)));

    SAFE_KERNEL_CALL((__calc_n__<<<gridDim, blockDim>>>(num)));    


    // SAFE_CALL(hipMemcpy((void*)b->next, (void*)temp, size * sizeof(double), hipMemcpyDeviceToHost));
    for (int i = 0; i < 6; ++i)
        SAFE_CALL(hipMemcpy((void*)b->edges[i], (void*)temp_edges[i], edge_sizes[i] * sizeof(double), hipMemcpyDeviceToHost));
}


__device__ double diff(unsigned int indx, int t) {
    unsigned int k = indx % nz_d        + sz_d;
    unsigned int j = indx / nz_d % ny_d + sy_d;
    unsigned int i = indx / nz_d / ny_d + sx_d;
    
    return abs(next_d[indx] - u_analytical_d(L_X_D,L_Y_D,L_Z_D, H_X_D*i, H_Y_D*j, H_Z_D*k, t*TAU_D));
}

__global__ void __err__(int t) {
    int gridOffsetX = blockDim.x * blockIdx.x;
    int gridOffsetY = blockDim.y * blockIdx.y;
    int gridOffsetZ = blockDim.z * blockIdx.z;

    // shift by s(.)_d, so i j k are indexing from the start of the block
    int i = threadIdx.x + gridOffsetX + sx_d;
    int j = threadIdx.y + gridOffsetY + sy_d;
    int k = threadIdx.z + gridOffsetZ + sz_d;

    // it shouldnt really happen, unless we use weird grid sizes
    if(i > ex_d || j > ey_d || k > ez_d)
        return;

    get(prev_d, i,j,k) = abs(get(next_d, i,j,k) - u_analytical_d(L_X_D,L_Y_D,L_Z_D, H_X_D*i, H_Y_D*j, H_Z_D*k, t*TAU_D));

}

extern double launch_err(Block * b) 
{
    // double * dev_out;
    // double out;
    // SAFE_CALL(hipMalloc((void**)&dev_out, sizeof(double)));

    // SAFE_KERNEL_CALL((__err__<<<1, REDUCTION_BLOCK_SIZE>>>(b->t, dev_out)));

    // SAFE_CALL(hipMemcpy(&out, dev_out, sizeof(double), hipMemcpyDeviceToHost));
    // SAFE_CALL(hipFree(dev_out));

    // return out;

    double * temp;
    double * res;
    double out;
    int size = b->nx * b->ny * b->nz;

    dim3 blockDim = dim3(THREAD_AXIS,THREAD_AXIS,THREAD_AXIS);
    int gridDimX = (b->nx - 1)/THREAD_AXIS + 1;
    int gridDimY = (b->ny - 1)/THREAD_AXIS + 1;
    int gridDimZ = (b->nz - 1)/THREAD_AXIS + 1;
    dim3 gridDim = dim3(gridDimX, gridDimY, gridDimZ);

    SAFE_CALL(hipMemcpyFromSymbol(&temp, HIP_SYMBOL(prev_d), sizeof(double *)));
    SAFE_KERNEL_CALL((__err__<<<gridDim, blockDim>>>(b->t)));
    

    res = thrust::max_element(thrust::device, temp, temp + size);
    SAFE_CALL(hipMemcpy(&out, res, sizeof(double), hipMemcpyDeviceToHost));

    return out;
}

extern void freeDevice() {
    double * temp;
    double * temp_edges[6];

    SAFE_CALL(hipMemcpyFromSymbol(&temp, HIP_SYMBOL(prev_d), sizeof(double *)));
    SAFE_CALL(hipFree(temp));

    SAFE_CALL(hipMemcpyFromSymbol(&temp, HIP_SYMBOL(curr_d), sizeof(double *)));
    SAFE_CALL(hipFree(temp));

    SAFE_CALL(hipMemcpyFromSymbol(&temp, HIP_SYMBOL(next_d), sizeof(double *)));
    SAFE_CALL(hipFree(temp));

    SAFE_CALL(hipMemcpyFromSymbol(&temp_edges, HIP_SYMBOL(edges_d), 6 * sizeof(double *)));
    for(int i=0; i<6; i++)
        SAFE_CALL(hipFree(temp_edges[i]));
}